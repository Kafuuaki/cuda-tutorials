
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void print_hello(void) {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {


    print_hello<<<1, 1>>>(); // 2 blocks, 4 threads per block
    hipDeviceSynchronize();

    return 0;
}