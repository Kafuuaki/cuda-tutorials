
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define Length 10000000

// why this code can not pass assertion?

void host_add(float *a, float *b, float *c, int n)
{
    for (int i = 0; i < n; i++)
    {
        c[i] = a[i] + b[i];
    }
}

__global__ void device_add(float *a, float *b, float *c)
{
    for (int i = 0; i < Length; i++)
    {
        c[i] = a[i] + b[i];
    }
}

void sum_assertion(float *a, float *b, float *c, int n)
{
    for (int i = 0; i < n; i++)
    {
        assert(c[i] == a[i] + b[i]);
    }

    printf("Sum Assertion passed\n");
}

int main()
{
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    // Allocate memory
    // should we use int here?
    // size_t actually
    size_t mem_size = Length * sizeof(float);

    a = (float *)malloc(mem_size);
    b = (float *)malloc(mem_size);
    c = (float *)malloc(mem_size);

    hipMalloc((void **)&d_a, mem_size);
    hipMalloc((void **)&d_b, mem_size);
    hipMalloc((void **)&d_c, mem_size);

    // Initialize the arrays
    for (int i = 0; i < Length; i++)
    {
        a[i] = i;
        b[i] = Length - i;
    }

    hipMemcpy(d_a, a, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, mem_size, hipMemcpyHostToDevice);

    // host_add(a, b, c, Length);

    device_add<<<1, Length>>>(d_a, d_b, d_c);

    // c is not ?
    hipMemcpy(c, d_c, mem_size, hipMemcpyDeviceToHost);

    // sum_assertion(a, b, c, Length);

    // for (int i = 0; i < Length; i++)
    // {
    //     printf("%f", c[i]);
    // }

    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}